
#include <hip/hip_runtime.h>
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef PADDLE_WITH_HETERPS
#include <algorithm>
#include <ctime>
#include <memory>
#include <numeric>
#include "paddle/fluid/framework/fleet/heter_ps/optimizer_conf.h"
#include "paddle/fluid/framework/fleet/ps_gpu_wrapper.h"
#include "paddle/fluid/framework/lod_tensor.h"
#include "paddle/fluid/platform/device/gpu/gpu_info.h"

namespace paddle {
namespace framework {

/*
__global__ void PullCopy(float** dest, const FeatureValue* src,
                         const int64_t* len, int hidden, int slot_num,
                         int total_len, uint64_t** keys) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[x - 1] : 0);
    if (*(keys[x] + y) == 0) {
      *(dest[x] + y * hidden) = 0;
      *(dest[x] + y * hidden + 1) = 0;
      *(dest[x] + y * hidden + 2) = 0;
    } else {
      *(dest[x] + y * hidden) = (src + i)->show;
      *(dest[x] + y * hidden + 1) = (src + i)->clk;
      *(dest[x] + y * hidden + 2) = (src + i)->lr;
    }
    if ((src + i)->mf_size == 0 || *(keys[x] + y) == 0) {
      for (int j = 0; j < hidden - 3; j++) {
        *(dest[x] + y * hidden + 3 + j) = 0;
      }
    } else {
      for (int j = 0; j < hidden - 3; j++) {
        *(dest[x] + y * hidden + 3 + j) = (src + i)->mf[1 + j];
      }
    }
  }
}

__global__ void PullCopy(float** dest, const FeatureValue* src,
                         const int64_t* len, int slot_num, int total_len,
                         uint64_t** keys, uint64_t max_val_size, int* gpu_dim) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[x - 1] : 0);
    int cur_dim = gpu_dim[x] - 3;
    FeatureValue* feature_value_ptr =
        (FeatureValue*)((char*)src + uint64_t(i) * uint64_t(max_val_size));
    
    int mf_dim = feature_value_ptr->mf_dim;
    mf_dim = gpu_dim[x] - 3;
    if (*(keys[x] + y) == 0) {
      *(dest[x] + y * (cur_dim + 3)) = 0;
      *(dest[x] + y * (cur_dim + 3) + 1) = 0;
      *(dest[x] + y * (cur_dim + 3) + 2) = 0;
    } else {
      *(dest[x] + y * (mf_dim + 3)) = feature_value_ptr->show;
      *(dest[x] + y * (mf_dim + 3) + 1) = feature_value_ptr->clk;
      *(dest[x] + y * (mf_dim + 3) + 2) = feature_value_ptr->lr;
    }
    if ((feature_value_ptr)->mf_size == 0 || *(keys[x] + y) == 0) {
      if (*(keys[x] + y) == 0) {
        for (int j = 0; j < cur_dim; j++) {
          *(dest[x] + y * (cur_dim + 3) + 3 + j) = 0;
        }
      } else {
        for (int j = 0; j < mf_dim; j++) {
          *(dest[x] + y * (mf_dim + 3) + 3 + j) = 0;
        }
      }
      
    } else {
      for (int j = 0; j < mf_dim; j++) {
        *(dest[x] + y * (mf_dim + 3) + 3 + j) = feature_value_ptr->mf[1 + j];
      }
    }
  }
}
*/

__global__ void CopyKeysKernel(uint64_t** src_keys, uint64_t* dest_total_keys,
                               const int64_t* len, int slot_num,
                               int total_len) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[x - 1] : 0);
    dest_total_keys[i] = src_keys[x][y];
  }
}

void PSGPUWrapper::CopyKeys(const paddle::platform::Place& place,
                            uint64_t** origin_keys, uint64_t* total_keys,
                            const int64_t* gpu_len, int slot_num,
                            int total_len) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(place))
                    ->stream();
  CopyKeysKernel<<<(total_len + 1024 - 1) / 1024, 1024, 0, stream>>>(
      origin_keys, total_keys, gpu_len, slot_num, total_len);
  cudaStreamSynchronize(stream);
}

void PSGPUWrapper::SetSparseSGD(float nonclk_coeff, float clk_coeff,
                                float min_bound, float max_bound,
                                float learning_rate, float initial_g2sum,
                                float initial_range) {
  optimizer_config_.set_sparse_sgd(nonclk_coeff,
                                  clk_coeff,
                                  min_bound,
                                  max_bound,
                                  learning_rate,
                                  initial_g2sum,
                                  initial_range);
}

void PSGPUWrapper::SetEmbedxSGD(float mf_create_thresholds,
                                float mf_learning_rate, float mf_initial_g2sum,
                                float mf_initial_range, float mf_min_bound,
                                float mf_max_bound) {
  optimizer_config_.set_embedx_sgd(mf_create_thresholds,
                                  mf_learning_rate,
                                  mf_initial_g2sum,
                                  mf_initial_range,
                                  mf_min_bound,
                                  mf_max_bound);
}

}  // end namespace framework
}  // end namespace paddle
#endif
